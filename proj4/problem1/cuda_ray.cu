#include <assert.h>
#include <hip/hip_runtime.h>


#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

#define CUDA 0
#define OPENMP 1
#define SPHERES 20 // 구체 개수

#define rnd( x ) (x * rand() / RAND_MAX)
#define INF 2e10f
#define DIM 2048

// 구체 정의
struct Sphere {
    float   r,b,g;
    float   radius;
    float   x,y,z;

		// ray hit
    __device__ float hit( float ox, float oy, float *n ) {
        float dx = ox - x;
        float dy = oy - y;
        if (dx*dx + dy*dy < radius*radius) {
            float dz = sqrtf( radius*radius - dx*dx - dy*dy );
            *n = dz / sqrtf( radius * radius );
            return dz + z;
        }
        return -INF;
    }
};

// 병렬화 부분
__global__ void kernel(Sphere* s, unsigned char* ptr)
{
	//픽셀 인덱스
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;
  
	int offset = x + y*DIM;
	float ox = (x - DIM/2);
	float oy = (y - DIM/2);

	//printf("x:%d, y:%d, ox:%f, oy:%f\n",x,y,ox,oy);

	float r=0, g=0, b=0;
	float   maxz = -INF;
	for(int i=0; i<SPHERES; i++) {
		float   n;
		float   t = s[i].hit( ox, oy, &n );
		if (t > maxz) {
			float fscale = n;
			r = s[i].r * fscale;
			g = s[i].g * fscale;
			b = s[i].b * fscale;
			maxz = t;
		} 
	}

	ptr[offset*4 + 0] = (int)(r * 255);
	ptr[offset*4 + 1] = (int)(g * 255);
	ptr[offset*4 + 2] = (int)(b * 255);
	ptr[offset*4 + 3] = 255;
}

void ppm_write(unsigned char* bitmap, int xdim,int ydim, FILE* fp)
{
	int i,x,y;
	fprintf(fp,"P3\n");
	fprintf(fp,"%d %d\n",xdim, ydim);
	fprintf(fp,"255\n");
	for (y=0;y<ydim;y++) {
		for (x=0;x<xdim;x++) {
			i=x+y*xdim;
			fprintf(fp,"%d %d %d ",bitmap[4*i],bitmap[4*i+1],bitmap[4*i+2]);
		}
		fprintf(fp,"\n");
	}
}

//메인코드
int main(int argc, char* argv[])
{
	int no_threads;
	int option;
	int x,y;
	unsigned char* bitmap;

	srand(time(NULL));

	if (argc!=3) {
		printf("> a.out [option] [filename.ppm]\n");
		printf("[option] 0: CUDA, 1~16: OpenMP using 1~16 threads\n");
		printf("for example, '> a.out 8 result.ppm' means executing OpenMP with 8 threads\n");
		exit(0);
	}
	FILE* fp = fopen(argv[2],"w");

	if (strcmp(argv[1],"0")==0) option=CUDA;
	else { 
		option=OPENMP;
		no_threads=atoi(argv[1]);
	}

	//구체 생성
	Sphere *temp_s = (Sphere*)malloc( sizeof(Sphere) * SPHERES );
	for (int i=0; i<SPHERES; i++) {
		temp_s[i].r = rnd( 1.0f );
		temp_s[i].g = rnd( 1.0f );
		temp_s[i].b = rnd( 1.0f );
		temp_s[i].x = rnd( 2000.0f ) - 1000;
		temp_s[i].y = rnd( 2000.0f ) - 1000;
		temp_s[i].z = rnd( 2000.0f ) - 1000;
		temp_s[i].radius = rnd( 200.0f ) + 40;
	}
	
	bitmap=(unsigned char*)malloc(sizeof(unsigned char)*DIM*DIM*4);
	
  //CUDA로 복사할 변수
	Sphere *dev_temp_s;
  unsigned char *dev_bitmap;
  
	//CUDA 메모리 할당
  hipMalloc((void**)&dev_temp_s, sizeof(Sphere) * SPHERES);
  hipMalloc((void**)&dev_bitmap, sizeof(unsigned char) * DIM * DIM * 4);

	//sphere: host -> device 복사
  hipMemcpy(dev_temp_s, temp_s, sizeof(Sphere) * SPHERES, hipMemcpyHostToDevice);

  
	int thread_num = 16;
  dim3 thread_per_block(thread_num, thread_num); //블록 당 쓰레드 수: 16x16
  dim3 block_num(DIM / thread_per_block.x, DIM / thread_per_block.y); //블록 수: 2048/16 x 2048/16

	clock_t start_time = clock(); // 시간 측정 시작
  kernel<<<block_num, thread_per_block>>>(dev_temp_s, dev_bitmap); //ray tracing 계산
	clock_t end_time = clock(); //시간 측정 끝
  
	clock_t diff_time = end_time - start_time;
	printf("CUDA ray tracing: %f sec. \n", (double)diff_time/CLOCKS_PER_SEC); // 시간 출력

	//tracing 결과: device -> host
  hipMemcpy(bitmap, dev_bitmap, sizeof(unsigned char) * DIM * DIM * 4, hipMemcpyDeviceToHost); 
	
  ppm_write(bitmap,DIM,DIM,fp);
	
	fclose(fp);
	free(bitmap);
	free(temp_s);

  hipFree(dev_temp_s);
  hipFree(dev_bitmap);

	return 0;
}
