#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/transform_reduce.h>
#include <thrust/execution_policy.h>
#include <iostream>

struct calculate_sum : public thrust::unary_function<long, double>
{
    double step;

    calculate_sum(double step) : step(step) {}

    __device__
    double operator()(const long& i) const
    {
        double x = (i + 0.5) * step;
        return 4.0 / (1.0 + x * x);
    }
};

int main()
{
    long num_steps = 1000000000;
    double step = 1.0 / static_cast<double>(num_steps);

    thrust::device_vector<long> indices(num_steps);
    thrust::sequence(indices.begin(), indices.end());

    clock_t start_time = clock();

    double sum = thrust::transform_reduce(thrust::device, indices.begin(), indices.end(),
                                          calculate_sum(step), 0.0, thrust::plus<double>());

    double pi = step * sum;

    clock_t end_time = clock();
    
    clock_t diff_time = end_time - start_time;
    printf("execution time: %f sec. \n", (double)diff_time/CLOCKS_PER_SEC);

    printf("pi = %.10lf\n", pi);

    return 0;
}
